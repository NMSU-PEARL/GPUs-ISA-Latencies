#include "hip/hip_runtime.h"
/*
** Author(s)      :  Yehia Arafa (yarafa@nmsu.edu)
** 
** File           :  pipeline.cu  
** 
** Description    :  Host (CPU) code to call each device (GPU) microbenchmark to compute
**                   the instructions latencies
** 
** Paper          :  Y. Arafa et al., "Low Overhead Instruction Latency Characterization
**                                     for NVIDIA GPGPUs," HPEC'19                                                  
*/

#include <stdio.h>
#include "device_kernels.cu"

 
int main(int argc, const char* argv[]){

    int n = 10;
    /* Host variable Declaration */
    int *c;
    /* Device variable Declaration */
    int  *d_c;
    /* Allocation of Host Variables */
    c = (int *)malloc(n * sizeof(int));
    /* Allocation of Device Variables */ 
    hipMalloc((void **)&d_c, n * sizeof(int));

    dim3 Dg = dim3(1); 
    dim3 Db = dim3(1);

// if (argc != 4){
//     printf("wrong number of argument\n"); 
//     exit(0);
// }
//====================== Kernel Start =========================
if(strcmp(argv[1],"Ovhd")==0){ 
    int clck = 0;
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/
        // PTX: 2 x mov
        //SASS: 2 x CS2R
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        clck = overhead;
        printf("PTX: 2 x mov.u32 rx clock;\nSASS: 2 x CS2R.32 Rx SR_CLOCKLO;\n--> number of cycles = %d\n",clck);

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        // PTX: 2 x mov
        //SASS: 2 x (CS2R + Mov) 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        clck = overhead;
        printf("PTX: 2 x mov.u32 rx clock;\nSASS: 2 x CS2R.32 Rx SR_CLOCKLO + 2 x mov;\n--> number of cycles = %d\n",clck);
        /*for the SASS clock overhead only:*/
        // Ovhd<<<Dg, Db>>>(d_c);
        // hipDeviceSynchronize();
        // hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        // int overhead = c[0];
        // Mov<<<Dg, Db>>>(d_c);
        // hipDeviceSynchronize();
        // hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        // int mov = (c[0] - overhead)/count_inst;
        // clck = overhead-(2*mov);
        // printf("SASS: \"2 x CS2R.32 Rx SR_CLOCKLO;\"\n--> number of cycles = %d\n",clck);
    }

}else if(strcmp(argv[1],"Nop")==0){
    int count_inst = 4;
    int nop = 0;
    if(atoi(argv[2]) == 3){ /*optimize-(opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Nop<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        nop = (c[0] - overhead)/count_inst;

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        //BRA.CONV + NOP, count_inst to get each SASS inst lat. = 4*2 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Nop<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        nop = (c[0] - overhead)/count_inst;
    }
    printf("PTX: bar.warp.sync;\nSASS: NOP\n--> number of cycles = %d\n",nop);

}else if(strcmp(argv[1],"BarSync")==0){
    // PTX: bar.sync
    //SASS: BAR.SYNC
    int count_inst = 4;
    int bar = 0;
    if(atoi(argv[2]) == 3){ /*optimize-(opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        BarSync<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        bar = (c[0] - overhead)/count_inst;
        printf("%d\n",c[0]);
        printf("PTX: bar.sync;\nSASS: BAR.SYNC\n--> number of cycles = %d\n",bar);

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        //WARPSYNC + BAR.SYNC
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        BarSync<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        bar = (c[0] - overhead)/count_inst;
        printf("PTX: bar.sync;\nSASS: WARPSYNC + BAR.SYNC\n--> number of cycles = %d\n",bar);
    }   

}else if(strcmp(argv[1],"MovSpec")==0){
    int count_inst = 2;
    int movspec = 0;
    // PTX: mov.u32 r1, tid.x
    //SASS: S2R
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        MovSpec<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        movspec = (c[0] - overhead - 0)/count_inst;
    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        MovSpec<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        movspec = (c[0] - overhead - (6*mov))/count_inst;
    }
    printf("PTX: mov.u32 r1, tid.x\nSASS: S2R\n--> number of cycles = %d\n",movspec);

}else if(strcmp(argv[1],"Mov")==0){
    // PTX: mov.u32
    //SASS: MOV
    if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        int count_inst = 6;
        int mov = 0;
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        mov = (c[0] - overhead)/count_inst;
        printf("PTX: mov.u32\nSASS: MOV\n--> number of cycles = %d\n",mov);
    }

}else if(strcmp(argv[1],"Cvt")==0){
    int count_inst = 3;
    int cvt = 0;
    // PTX: cvt
    //SASS: F2I
    if(atoi(argv[2]) == 3){ /*optimize-(opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Cvt<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        cvt = (c[0] - overhead)/count_inst;

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        Cvt<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        cvt = (c[0] - overhead-(3*mov))/count_inst;
    }
    printf("PTX: cvt.rzi.s32.f32\nSASS: F2I.TRUNC.NTZ\n--> number of cycles = %d\n",cvt);

}else if(strcmp(argv[1],"Add")==0){
    int count_inst = 3;
    int add = 0;
    // PTX: add.u32
    //SASS: IADD
    if(atoi(argv[2]) == 3){ /*optimize-(opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Add<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        add = (c[0] - overhead)/count_inst;

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        Add<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        add = (c[0] - overhead-(3*mov))/count_inst;
    }
    printf("PTX: add.u32\nSASS: IADD\n--> number of cycles = %d\n",add);

}else if(strcmp(argv[1],"Mul")==0){
    int count_inst = 3;
    int mul = 0;
    // PTX: mul.u32
    //SASS: IMul
    if(atoi(argv[2]) == 3){ /*optimize-(opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mul<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        mul = (c[0] - overhead-4)/count_inst;

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        Mul<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        mul = (c[0] - overhead-(3*mov))/count_inst;
    }
    printf("PTX: mul.lo.u32\nSASS: IMUL\n--> number of cycles = %d\n",mul);

}else if(strcmp(argv[1],"Div")==0){
    int count_inst = 2;
    int div = 0;
    Ovhd<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    int overhead = c[0];
    Div<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    div = (c[0] - overhead)/count_inst;
    
    printf("PTX: div.s32\nSASS: [emulated]\n--> number of cycles = %d\n",div);

}else if(strcmp(argv[1],"DivU")==0){
    int count_inst = 2;
    int div = 0;
    Ovhd<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    int overhead = c[0];
    DivU<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    div = (c[0] - overhead)/count_inst;

    printf("PTX: div.u32\nSASS: [emulated]\n--> number of cycles = %d\n",div);

}else if(strcmp(argv[1],"Rem")==0){
    int count_inst = 2;
    int rem = 0; 
    Ovhd<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    int overhead = c[0];
    Rem<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    rem = (c[0] - overhead)/count_inst;

    printf("PTX: rem.s32\nSASS: [emulated]\n--> number of cycles = %d\n",rem);

}else if(strcmp(argv[1],"RemU")==0){
    int count_inst = 2;
    int rem = 0;
    Ovhd<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    int overhead = c[0];
    RemU<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    rem = (c[0] - overhead)/count_inst;
    
    printf("PTX: rem.u32\nSASS: [emulated]\n--> number of cycles = %d\n",rem);

}else if(strcmp(argv[1],"Mul24Lo")==0){
    int count_inst = 2;
    int mul = 0;
    if(atoi(argv[2]) == 3){ /*optimize-(opt -O3)*/ 
        // PTX: mul24.lo.u32
        //SASS: SGXT.U32 + IMAD, count_inst to get each SASS inst lat. = 2*2
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mul24Lo<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        mul = (c[0] - overhead)/count_inst;
        printf("PTX: mul24.lo.u32\nSASS: [emulated] SGXT.U32 + IMAD\n--> number of cycles = %d\n",mul);

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        // PTX: mul24.lo.u32
        //SASS: SHF.R.U32 + SGXT.U32 + IMAD + Mov, count_inst to get each SASS inst lat. = 2*4
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mul24Lo<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        mul = (c[0] - overhead)/count_inst;
        printf("PTX: mul24.lo.u32\nSASS: [emulated] SHF.R.U32 + SGXT.U32 + IMAD + Mov\n--> number of cycles = %d\n",mul);
    }

}else if(strcmp(argv[1],"Mul24Hi")==0){
    int count_inst = 2;
    int mul = 0;
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/ 
        // PTX: mul24.hi.u32
        //SASS: SGXT.U32 + IMAD.WIDE + SHF + PRMT, count_inst to get SASS = 2*3
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mul24Hi<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        mul = (c[0] - overhead)/count_inst;
        printf("PTX: mul24.hi.u32\nSASS: [emulated] SGXT.U32 + IMAD.WIDE.U32 + SHF.R.U32.HI + PRMT\n--> number of cycles = %d\n",mul);

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        // PTX: mul24.hi.u32
        //SASS: SGXT.U32 + IMAD.WIDE + SHF + PRMT + Mov
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mul24Hi<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        mul = (c[0] - overhead)/count_inst;
        printf("PTX: mul24.hi.u32\nSASS: [emulated] SGXT.U32 + IMAD.WIDE + SHF.R.U32.HI + LOP3.LUT + Mov\n--> number of cycles = %d\n",mul);
    }

}else if(strcmp(argv[1],"Popc")==0){
    int count_inst = 3;
    int popc = 0;
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/ 
        // PTX: popc.b32
        //SASS: POPC
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Popc<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        popc = (c[0] - overhead)/count_inst;
        printf("PTX: popc.b32\nSASS: POPC\n--> number of cycles = %d\n",popc);
    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        // PTX: popc.b32
        //SASS: POPC + 6 x LOP3.LUT + 3 x MOV, count_inst to get each SASS inst lat. = 9
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        Popc<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        popc = (c[0] - overhead-(3*mov))/count_inst;
        printf("PTX: popc.b32\nSASS: POPC + LOP3.LUT + MOV\n--> number of cycles = %d\n",popc);
    }
    
}else if(strcmp(argv[1],"Sad")==0){
    int count_inst = 3;
    int sad = 0;
    // PTX: sad.b32
    //SASS: VABSDIFF
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Sad<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        sad = (c[0] - overhead)/count_inst;
    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        Sad<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        sad = (c[0] - overhead-(4*mov))/count_inst;
    }
    printf("PTX: sad.u32\nSASS: VABSDIFF\n--> number of cycles = %d\n",sad);

}else if(strcmp(argv[1],"Clz")==0){
    int count_inst = 4;
    int clz = 0;
    // PTX: clz.b32
    //SASS: FLO.U32 + IADD
    if(atoi(argv[2]) == 3){ /*optimizer (opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Clz<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        clz = (c[0] - overhead)/count_inst;
    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        Clz<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        clz = (c[0] - overhead-(2*mov))/count_inst;
    }
    printf("PTX: clz.b32\nSASS: FLO.U32 + IADD\n--> number of cycles = %d\n",clz);

}else if(strcmp(argv[1],"Bfind")==0){
    int count_inst = 3;
    int bfind = 0;
    // PTX: bfind.u32
    //SASS: FLO.U32
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Bfind<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        bfind = (c[0] - overhead)/count_inst;
    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        Bfind<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        bfind = (c[0] - overhead-(3*mov))/count_inst;
    }
    printf("PTX: bfind.u32\nSASS: FLO.U32\n--> number of cycles = %d\n",bfind);

}else if(strcmp(argv[1],"Brev")==0){
    int count_inst = 4;
    int brev = 0;
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/
        // PTX: brev.u32
        //SASS: BREV + SGXT.U32 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Brev<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        brev = (c[0] - overhead)/count_inst;
        printf("PTX: brev.b32 \nSASS: BREV + SGXT.U32\n--> number of cycles = %d\n",brev);
    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        // PTX: brev.u32
        //SASS: BREV + SHF.R.U32.HI + SGXT.U32
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        Brev<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        brev = (c[0] - overhead-(2*mov))/2;
        printf("PTX: brev.b32 \nSASS: BREV + SHF.R.U32.HI + SGXT.U32\n--> number of cycles = %d\n",brev);
    }
    
}else if(strcmp(argv[1],"Bfe")==0){
    int count_inst = 4;
    int bfe = 0;
    Ovhd<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    int overhead = c[0];
    Bfe<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    bfe = (c[0] - overhead)/count_inst;
    printf("PTX: bfe.u32\nSASS: [emulated]\n--> number of cycles = %d\n",bfe);

}else if(strcmp(argv[1],"And")==0){
    int count_inst = 3;
    int annd = 0;
    // PTX: and.u32
    //SASS: LOP3
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        And<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        annd = (c[0] - overhead)/count_inst;
    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        And<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        annd = (c[0] - overhead-(3*mov))/count_inst;
    }
    printf("PTX: and.b32\nSASS: LOP3.LUT\n--> number of cycles = %d\n",annd);

}else if(strcmp(argv[1],"Cnot")==0){ 
    int count_inst = 2;
    int cnot = 0;
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/
        // PTX: cnot.b32
        //SASS: SETP + SEL 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Cnot<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        cnot = (c[0] - overhead)/count_inst;
        printf("PTX: cnot.b32\nSASS: SETP + SEL\n--> number of cycles = %d\n",cnot);
    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        // PTX: cnot.b32
        //SASS: SETP + SEL + IADD3 + 6 x mov
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        Cnot<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        cnot = (c[0] - overhead-(2*mov))/count_inst;
        printf("PTX: cnot.b32\nSASS: SETP + SEL + IADD3 + 2 x Mov\n--> number of cycles = %d\n",cnot);
    }
    
}else if(strcmp(argv[1],"MAddc")==0){
    int count_inst = 3;
    int add = 0;
    // PTX: addc.u32
    //SASS: IADD3.X
    if(atoi(argv[2]) == 3){ /*optimize-(opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        MAddc<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        add = (c[0] - overhead)/count_inst;
    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        MAddc<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        add = (c[0] - overhead-(3*mov))/count_inst;
    }
    printf("PTX: addc.u32\nSASS: IADD3.X\n--> number of cycles = %d\n",add);

}else if(strcmp(argv[1],"Setp")==0){ //count_inst = 800
    int count_inst = 800;
    int setp = 0;
    if(atoi(argv[2]) == 3){ /*optimize-(opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Setp<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        setp = (c[0] - (overhead*10000))/count_inst;

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/ 
       count_inst = 10000;
        // SASS: ISETP + PLOP3, count_inst to get SASS inst. = 10000*2 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Setp<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        setp = (c[0] - overhead*10000)/count_inst;
    }
    printf("PTX: setp.ne.s32\nSASS: ISETP.NE.AND\n--> number of cycles = %d\n",setp);

}else if(strcmp(argv[1],"FAdd")==0){ 
    int count_inst = 3;
    int add = 0;
    // PTX: add.f32
    //SASS: FADD
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        FAdd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        add = (c[0] - overhead)/count_inst;

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        FAdd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        add = (c[0] - overhead-(3*mov))/count_inst;
    }
    printf("PTX: add.f32\nSASS: FADD\n--> number of cycles = %d\n",add);

}else if(strcmp(argv[1],"FMul")==0){
    int count_inst = 3;
    int mul = 0;
    // PTX: mul.f32
    //SASS: FMUL
    if(atoi(argv[2]) == 3){ /*optimize-(opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        FMul<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        mul = (c[0] - overhead)/count_inst;

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        FMul<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        mul = (c[0] - overhead-(3*mov))/count_inst;
    }
    printf("PTX: mul.rn.f32\nSASS: FMUL\n--> number of cycles = %d\n",mul);

}else if(strcmp(argv[1],"FFMa")==0){
    int count_inst = 3;
    int fma = 0;
    // PTX: fma.f32
    //SASS: FFMA
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        FFMa<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        fma = (c[0] - overhead)/count_inst;

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        FFMa<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        fma = (c[0] - overhead-(3*mov))/count_inst;
    }
    printf("PTX: fma.rn.f32\nSASS: FFMA\n--> number of cycles = %d\n",fma);

}else if(strcmp(argv[1],"FDiv")==0){
    int count_inst = 2;
    int div = 0;
    Ovhd<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    int overhead = c[0];
    FDiv<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    div = (c[0] - overhead)/count_inst;
    
    printf("PTX: div.rn.f32\nSASS: [emulated]\n--> number of cycles = %d\n",div);

}else if(strcmp(argv[1],"DFAdd")==0){
    int count_inst = 2;
    int add = 0;
    // PTX: add.rn.f64
    //SASS: DADD
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        DFAdd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        add = (c[0] - overhead)/count_inst;

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        DFAdd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        add = (c[0] - overhead-(2*mov))/count_inst;
    }
    printf("PTX: add.rn.f64\nSASS: DADD\n--> number of cycles = %d\n",add);

}else if(strcmp(argv[1],"DFMul")==0){
    int count_inst = 2;
    int mul = 0;
    // PTX: mul.rn.f64
    //SASS: DMUL
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        DFMul<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        mul = (c[0] - overhead)/count_inst;

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        DFMul<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        mul = (c[0] - overhead-(2*mov))/count_inst;
    }
    printf("PTX: mul.rn.f64\nSASS: DMUL\n--> number of cycles = %d\n",mul);

}else if(strcmp(argv[1],"DFFMA")==0){
    int count_inst = 2;
    int fma = 0;
    // PTX: fma.rn.f64
    //SASS: DFMA
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        DFFMA<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        fma = (c[0] - overhead)/count_inst;

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        DFFMA<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        fma = (c[0] - overhead-(2*mov))/count_inst;
    }
    printf("PTX: fma.rn.f64\nSASS: DFMA\n--> number of cycles = %d\n",fma);

}else if(strcmp(argv[1],"DFDiv")==0){
    int count_inst = 2;
    int div = 0;
    Ovhd<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    int overhead = c[0];
    DFDiv<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    div = (c[0] - overhead)/count_inst;

    printf("PTX: div.rn.f64\nSASS: [emulated]\n--> number of cycles = %d\n",div);

}else if(strcmp(argv[1],"HFAdd")==0){
    int count_inst = 3;
    int add = 0;
    // PTX: add.f16
    //SASS: HADD2
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        HFAdd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        add = (c[0] - overhead)/count_inst;

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        HFAdd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        add = (c[0] - overhead-(3*mov))/count_inst;
    }
    printf("PTX: add.f16\nSASS: HADD\n--> number of cycles = %d\n",add);

}else if(strcmp(argv[1],"HFMul")==0){
    int count_inst = 3;
    int mul = 0;
    // PTX: mul.f16
    //SASS: HMUL
    if(atoi(argv[2]) == 3){ /*optimize(opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        HFMul<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        mul = (c[0] - overhead)/count_inst;

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        HFMul<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        mul = (c[0] - overhead-(3*mov))/count_inst;
    }
    printf("PTX: mul.rn.f16\nSASS: HMUL\n--> number of cycles = %d\n",mul);

}else if(strcmp(argv[1],"HFFMa")==0){
    int count_inst = 3;
    int fma = 0;
    // PTX: fma.f16
    //SASS: HFMA
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        HFFMa<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        fma = (c[0] - overhead)/count_inst;

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        HFFMa<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        fma = (c[0] - overhead-(3*mov))/count_inst;
    }
    printf("PTX: fma.rn.f16\nSASS: HFMA\n--> number of cycles = %d\n",fma);

}else if(strcmp(argv[1],"Rcp")==0){
    int count_inst = 3;
    int rcp = 0;
    Ovhd<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    int overhead = c[0];
    Rcp<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    rcp = (c[0] - overhead)/count_inst;

    printf("PTX: rcp.rn.f32\nSASS: [multiple insts including MUFU.RCP]\n--> number of cycles = %d\n",rcp);

}else if(strcmp(argv[1],"FastRcp")==0){
    int count_inst = 3;
    int rcp = 0;
    Ovhd<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    int overhead = c[0];
    FastRcp<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    rcp = (c[0] - overhead)/count_inst;

    printf("PTX: rcp.approx.f32\nSASS: [multiple insts including MUFU.RCP]\n--> number of cycles = %d\n",rcp);

}else if(strcmp(argv[1],"DRcp")==0){
    int count_inst = 2;
    int rcp = 0;
    Ovhd<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    int overhead = c[0];
    DRcp<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    rcp = (c[0] - overhead)/count_inst;

    printf("PTX: rcp.rn.f64\nSASS: [multiple insts including MUFU.RCP64H]\n--> number of cycles = %d\n",rcp);

}else if(strcmp(argv[1],"Sqrt")==0){
    int count_inst = 3;
    int sqrt = 0;
    Ovhd<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    int overhead = c[0];
    Sqrt<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    sqrt = (c[0] - overhead)/count_inst;

    printf("PTX: sqrt.rn.f32\nSASS: [multiple insts including MUFU.RSQ]\n--> number of cycles = %d\n",sqrt);

}else if(strcmp(argv[1],"DSqrt")==0){
    int count_inst = 2;
    int sqrt = 0;
    Ovhd<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    int overhead = c[0];
    DSqrt<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    sqrt = (c[0] - overhead)/count_inst;

    printf("PTX: sqrt.rn.f64\nSASS: [multiple insts including MUFU.RSQ64]\n--> number of cycles = %d\n",sqrt);

}else if(strcmp(argv[1],"FastSqrt")==0){
    int count_inst = 3;
    int sqrt = 0;
    Ovhd<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    int overhead = c[0];
    FastSqrt<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    sqrt = (c[0] - overhead)/count_inst;

    printf("PTX: sqrt.approx.f32\nSASS: [multiple insts including MUFU.SQRT]\n--> number of cycles = %d\n",sqrt);

}else if(strcmp(argv[1],"Rsqrt")==0){
    int count_inst = 3;
    int sqrt = 0;
    Ovhd<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    int overhead = c[0];
    Rsqrt<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    sqrt = (c[0] - overhead)/count_inst;

    printf("PTX: rsqrt.approx.f32\nSASS: [multiple insts including MUFU.RSQ]\n--> number of cycles = %d\n",sqrt);

}else if(strcmp(argv[1],"FastDRsqrt")==0){
    int count_inst = 4;
    int sqrt = 0;
    // PTX: rsqrt.approx.ftz.f64
    //SASS: MUFU.RSQ64H
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        FastDRsqrt<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        sqrt = (c[0] - overhead)/count_inst;

    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        FastDRsqrt<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        sqrt = (c[0] - overhead-(12*mov))/count_inst;
    }
    printf("PTX: rsqrt.approx.ftz.f64\nSASS: MUFU.RSQ64H\n--> number of cycles = %d\n",sqrt);

}else if(strcmp(argv[1],"Sin")==0){
    int count_inst = 1;
    int sin = 0;
    // PTX: sin.approx.f32
    //SASS: FMUL + MUFU.SIN, count_inst to get each SASS inst lat. = 2 * 2
    if(atoi(argv[2]) == 3){ /*optimize (opt -O3)*/ 
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Sin<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        sin = (c[0] - overhead)/count_inst;
    }else if(atoi(argv[2]) == 0){ /*non-optimize (nonOpt -O0)*/
        Ovhd<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int overhead = c[0];
        Mov<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        int mov = (c[0] - overhead)/6;
        Sin<<<Dg, Db>>>(d_c);
        hipDeviceSynchronize();
        hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
        sin = (c[0] - overhead-(2*mov))/count_inst;
    }
    printf("PTX: sin.approx.f32\nSASS: FMUL + MUFU.SIN\n--> number of cycles = %d\n",sin);

}else if(strcmp(argv[1],"Ex2")==0){
    int count_inst = 3;
    int ex = 0;
    // PTX: ex2.approx.f32
    //SASS: FSTEP + FMUL + MUFU.LG2 + FADD, count_inst to get each SASS inst lat. = 3 * 4
    Ovhd<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    int overhead = c[0];
    Ex2<<<Dg, Db>>>(d_c);
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
    ex = (c[0] - overhead)/count_inst;
    printf("PTX: ex2.approx.f32\nSASS: FSTEP + FMUL + MUFU.EX2 + FMUL\n--> number of cycles = %d\n",ex);

}else { 
    printf("Wrong Instruction\n"); 
    exit(0);
}

    /* Free Device Memory */
    hipFree(d_c);
    /* Free Host Memory */
    free(c);
    

    return 0;
}