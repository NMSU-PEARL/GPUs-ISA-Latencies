
#include <hip/hip_runtime.h>
/*
** Author(s)      :  Yehia Arafa (yarafa@nmsu.edu)
** 
** File           :  device_kernels.cu  
** 
** Description    :  device kernels declarations
** 
** Paper          :  Y. Arafa et al., "Low Overhead Instruction Latency Characterization
**                                     for NVIDIA GPGPUs," HPEC'19                                                  
*/


/* Miscellaneous & Synchronization Instruction */
__global__ void Ovhd(int *c) {}
__global__ void Nop(int *c) {}
__global__ void BarSync(int *c) {}

/* Movement Instructions */
__global__ void MovSpec(int *c) {}
__global__ void Mov(int *c) {}

/* Conversion Instructions */
__global__ void Cvt(int *c) {}

/* Integer Instructions */
__global__ void Add(int *c) {}
__global__ void Mul(int *c) {}
__global__ void Div(int *c) {} //PTX only [SASS emulated]
__global__ void DivU(int *c) {} //PTX only [SASS emulated]
__global__ void Rem(int *c) {} //PTX only [SASS emulated]
__global__ void RemU(int *c) {} //PTX only [SASS emulated]
__global__ void Mul24Lo(int *c) {}
__global__ void Mul24Hi(int *c) {}
__global__ void Popc(int *c) {}
__global__ void Sad(int *c) {}
__global__ void Clz(int *c) {}
__global__ void Bfind(int *c) {}
__global__ void Brev(int *c) {}
__global__ void Bfe(int *c) {} //PTX only [SASS emulated]
    //--Logic Instructions--//
__global__ void And(int *c) {} //=copysign PTX instruction
__global__ void Cnot(int *c) {} 
    //--Multi Percision Instructions--//
__global__ void MAddc(int *c) {}
    //--Comparison and Selection Instructions--//
__global__ void Setp(int *c) {}

/* Floating Point Instructions */
    //--FP32 Single Precision Instructions--//
__global__ void FAdd(int *c) {}
__global__ void FMul(int *c) {}
__global__ void FFMa(int *c) {}
__global__ void FDiv(int *c) {} //PTX only [SASS emulated]
    //--FP64 Double Precision Instructions--//
__global__ void DFAdd(int *c) {}
__global__ void DFMul(int *c) {}
__global__ void DFFMA(int *c) {}
__global__ void DFDiv(int *c) {} //PTX only [SASS emulated]
    //--FP16 Half Precision Instructions--//
__global__ void HFAdd(int *c) {}
__global__ void HFMul(int *c) {}
__global__ void HFFMa(int *c) {}

/* SFU Special Instructions */
__global__ void Rcp(int *c) {} //PTX only [SASS emulated]
__global__ void FastRcp(int *c) {} //PTX Only [SASS emulated]
__global__ void DRcp(int *c) {} //PTX only [SASS emulated]
__global__ void Sqrt(int *c) {} //PTX only [SASS emulated]
__global__ void FastSqrt(int *c) {} //PTX only [SASS emulated]
__global__ void DSqrt(int *c) {} //PTX only [SASS emulated]
__global__ void Rsqrt(int *c) {} //PTX only [SASS emulated]
__global__ void FastDRsqrt(int *c) {} //=FastDRcp in PTX (or MUFU.RCP64H in SASS) instruction 
__global__ void Sin(int *c) {}
__global__ void Ex2(int *c) {} //= Lg2 in PTX (MUFU.LG2 in SASS). PTX only [SASS emulated]